#include "hip/hip_runtime.h"
#define gpuErrorCheck(ans)                    \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
#include <stdlib.h>
#include <algorithm>
#include <cmath>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include "gpu.cuh"
#include "defines.h"
#include "hip/hip_runtime_api.h"

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

__device__ bool isInRowGpu(int row, int number, int *board)
{
    for (int col = 0; col < BOARD_SIZE; col++)
    {
        if (board[row * BOARD_SIZE + col] == number)
            return true;
    }

    return false;
}

__device__ bool isInColumnGpu(int column, int number, int *boards)
{
    for (int row = 0; row < BOARD_SIZE; row++)
    {
        if (boards[row * BOARD_SIZE + column] == number)
            return true;
    }

    return false;
}

__device__ bool isInBoxGpu(int boxStartRow, int boxStartColumn, int number, int *board)
{
    for (int row = 0; row < BOX_SIZE; row++)
    {
        for (int col = 0; col < BOX_SIZE; col++)
        {
            if (board[boxStartRow * BOARD_SIZE + row * BOARD_SIZE + boxStartColumn + col] == number)
                return true;
        }
    }

    return false;
}

__device__ bool isValidPlace(int row, int column, int number, int *boards, int idx)
{
    int *board = boards + idx * (BOARD_SIZE * BOARD_SIZE);
    return !isInRowGpu(row, number, board) && !isInColumnGpu(column, number, board) && !isInBoxGpu(row - row % BOX_SIZE, column - column % BOX_SIZE, number, board);
}

__device__ void clearVisited(bool *visited)
{
    for (int i = 0; i < BOARD_SIZE; i++)
    {
        visited[i] = false;
    }
}

__device__ bool isRowValid(int row, const int *board)
{
    bool visited[BOARD_SIZE] = {false};

    for (int i = 0; i < BOARD_SIZE; i++)
    {
        int num = board[row * BOARD_SIZE + i];

        if (num != 0)
        {
            if (visited[num - 1])
            {
                return false;
            }
            else
            {
                visited[num - 1] = true;
            }
        }
    }

    return true;
}

__device__ bool isColumnValid(int column, const int *board)
{
    bool visited[BOARD_SIZE] = {false};

    for (int i = 0; i < BOARD_SIZE; i++)
    {
        int num = board[i * BOARD_SIZE + column];

        if (num != 0)
        {
            if (visited[num - 1])
            {
                return false;
            }
            else
            {
                visited[num - 1] = true;
            }
        }
    }

    return true;
}

__device__ bool isBoxValid(int rowIdx, int columnIdx, const int *board)
{
    bool visited[BOARD_SIZE] = {false};
    for (int i = 0; i < BOX_SIZE; i++)
    {
        for (int j = 0; j < BOX_SIZE; j++)
        {
            int num = board[(rowIdx * BOX_SIZE + i) * BOARD_SIZE + (columnIdx * BOX_SIZE + j)];

            if (num != 0)
            {
                if (visited[num - 1])
                {
                    return false;
                }
                else
                {
                    visited[num - 1] = true;
                }
            }
        }
    }

    return true;
}

__device__ bool isBoardValid(const int *board, int idx)
{
    int row = idx / BOARD_SIZE;
    int column = idx % BOARD_SIZE;

    int rowIdx = row / BOX_SIZE;
    int columnIdx = column / BOX_SIZE;

    if ((board[idx] < 1) || (board[idx] > 9))
    {
        return false;
    }

    return isRowValid(row, board) && isColumnValid(column, board) && isBoxValid(rowIdx, columnIdx, board);
}

// each thread generates new boards from array of current boards with valid inserts
__global__ void bfs(
    int *currentBoards,
    unsigned int currentBoardsCount,
    int *nextBoards,
    unsigned int *nextBoardIdx,
    int *emptySpaces,
    int *emptySpacesCounts)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int startIdx = idx * BOARD_SIZE * BOARD_SIZE;
    int endIdx = (idx * BOARD_SIZE * BOARD_SIZE) + BOARD_SIZE * BOARD_SIZE;
    bool foundEmptyPlace = false;

    // finish if ran out of boards for current thread
    while (idx < currentBoardsCount)
    {
        foundEmptyPlace = false;

        for (int i = startIdx; i < endIdx && !foundEmptyPlace; i++)
        {
            if (currentBoards[i] == 0)
            {
                int shift = i - BOARD_SIZE * BOARD_SIZE * idx;

                // row and column for current empty place
                int currentRow = shift / BOARD_SIZE;
                int currentColumn = shift % BOARD_SIZE;
                foundEmptyPlace = true;

                for (int possibleNumber = 1; possibleNumber <= 9; possibleNumber++)
                {
                    if (isValidPlace(currentRow, currentColumn, possibleNumber, currentBoards, idx))
                    {
                        int nextIdx = atomicAdd(nextBoardIdx, 1);
                        int emptyCount = 0;
                        foundEmptyPlace = true;

                        for (int row = 0; row < BOARD_SIZE; row++)
                        {
                            for (int column = 0; column < BOARD_SIZE; column++)
                            {
                                int nextBoardIdx = nextIdx * (BOARD_SIZE * BOARD_SIZE) + row * BOARD_SIZE + column;
                                int currentBoardIdx = idx * (BOARD_SIZE * BOARD_SIZE) + row * BOARD_SIZE + column;

                                // copy currentBoard to nextBoard
                                nextBoards[nextBoardIdx] = currentBoards[currentBoardIdx];
                                if (currentBoards[currentBoardIdx] == 0 && !(row == currentRow && column == currentColumn))
                                {
                                    emptySpaces[emptyCount + BOARD_SIZE * BOARD_SIZE * nextIdx] = row * BOARD_SIZE + column;
                                    emptyCount++;
                                }
                            }
                        }
                        emptySpacesCounts[nextIdx] = emptyCount;
                        nextBoards[nextIdx * (BOARD_SIZE * BOARD_SIZE) + currentRow * BOARD_SIZE + currentColumn] = possibleNumber;
                    }
                }
            }
        }
        // if idx is less than currentBoardCount find next board in currentBoards
        idx += blockDim.x * gridDim.x;
    }
}

__global__ void dfs(int *currentBoards,
                    unsigned int currentBoardsCount,
                    int *emptySpaces,
                    int *emptySpacesCounts,
                    int *finished,
                    int *result)
{

    int index = blockDim.x * blockIdx.x + threadIdx.x;

    int *currentBoard;
    int *currentEmptySpaces;
    int currentEmptySpacesCount;

    while ((*finished == 0) && (index < currentBoardsCount))
    {
        int emptyIndex = 0;

        currentBoard = currentBoards + index * BOARD_SIZE * BOARD_SIZE;
        currentEmptySpaces = emptySpaces + index * BOARD_SIZE * BOARD_SIZE;
        currentEmptySpacesCount = emptySpacesCounts[index];

        // check specific board
        while ((emptyIndex >= 0) && (emptyIndex < currentEmptySpacesCount))
        {
            currentBoard[currentEmptySpaces[emptyIndex]]++;

            if (isBoardValid(currentBoard, currentEmptySpaces[emptyIndex]))
            {
                emptyIndex++;
            }
            // if board is invalid and we checked all possibilities mark current place in board as empty and go back in empty indexes
            else if (currentBoard[currentEmptySpaces[emptyIndex]] >= 9)
            {
                currentBoard[currentEmptySpaces[emptyIndex]] = 0;
                emptyIndex--;
            }
        }

        // if all empty spaces have been filled solution was found
        if (emptyIndex == currentEmptySpacesCount)
        {
            *finished = 1;

            for (int i = 0; i < BOARD_SIZE * BOARD_SIZE; i++)
            {
                result[i] = currentBoard[i];
            }
        }

        // go to next board
        index += gridDim.x * blockDim.x;
    }
}

void swapBoards(int *&currentBoards, int *&nextBoards)
{
    int *temp = currentBoards;
    currentBoards = nextBoards;
    nextBoards = temp;
}

float solveWithGpu(int *sudoku, int *result)
{
    clock_t executionStart, executionEnd, copyingStart, copyingEnd;
    // boards from which next set of boards will be generated
    int *currentBoards;
    // set of boards generated from currentBoards
    int *nextBoards;
    // index of next free space for generated board in nextBoards
    unsigned int *boardIdx;
    // flag if solution was found
    int *finished;
    // solved sudoku
    int *gpuResult;
    // number of boards in currentBoards
    unsigned int boardCount = 1;
    // indexes of empty spaces in specific board
    int *emptySpaces;
    // number of empty spaces in specific board
    int *emptySpacesCounts;
    // max size for all boards
    const double maxBoardsSize = BOARD_SIZE * BOARD_SIZE * MAX_BOARDS;
    float executionTime;
    float dataCopyingTime;

    std::copy(sudoku, sudoku + BOARD_SIZE * BOARD_SIZE, result);

    gpuErrorCheck(hipFree(0));
    gpuErrorCheck(hipSetDevice(0));

    gpuErrorCheck(hipMalloc(&nextBoards, maxBoardsSize * sizeof(int)));
    gpuErrorCheck(hipMalloc(&currentBoards, maxBoardsSize * sizeof(int)));
    gpuErrorCheck(hipMalloc(&boardIdx, sizeof(unsigned int)));
    gpuErrorCheck(hipMalloc(&finished, sizeof(int)));
    gpuErrorCheck(hipMalloc(&gpuResult, BOARD_SIZE * BOARD_SIZE * sizeof(int)));
    gpuErrorCheck(hipMalloc(&emptySpaces, maxBoardsSize * sizeof(int)));
    gpuErrorCheck(hipMalloc(&emptySpacesCounts, (maxBoardsSize / (BOARD_SIZE * BOARD_SIZE) + 1) * sizeof(int)));

    gpuErrorCheck(hipMemset(boardIdx, 0, sizeof(int)));
    gpuErrorCheck(hipMemset(finished, 0, sizeof(int)));
    gpuErrorCheck(hipMemset(nextBoards, 0, maxBoardsSize * sizeof(int)));
    gpuErrorCheck(hipMemset(currentBoards, 0, maxBoardsSize * sizeof(int)));
    gpuErrorCheck(hipMemset(gpuResult, 0, BOARD_SIZE * BOARD_SIZE * sizeof(int)));

    copyingStart = clock();
    gpuErrorCheck(hipMemcpy(currentBoards, result, BOARD_SIZE * BOARD_SIZE * sizeof(int), hipMemcpyHostToDevice));
    copyingEnd = clock();

    dataCopyingTime = ((float)(copyingEnd - copyingStart)) / (CLOCKS_PER_SEC / 1000);

    executionStart = clock();

    for (int i = 0; i < ITERATIONS_COUNT; i++)
    {
        // start adding boards from 0 index in nextBoards
        gpuErrorCheck(hipMemset(boardIdx, 0, sizeof(unsigned int)));

        // find next array of possible boards from currentBoards, and add them to nextBoards
        bfs<<<BLOCKS, THREADS>>>(
            currentBoards,
            boardCount,
            nextBoards,
            boardIdx,
            emptySpaces,
            emptySpacesCounts);

        // use nextBoards as currentBoards in next iteration
        swapBoards(currentBoards, nextBoards);

        copyingStart = clock();
        // get number of possible generated boards
        gpuErrorCheck(hipMemcpy(&boardCount, boardIdx, sizeof(unsigned int), hipMemcpyDeviceToHost));
        copyingEnd = clock();
        dataCopyingTime += ((float)(copyingEnd - copyingStart)) / (CLOCKS_PER_SEC / 1000);
    }
    dfs<<<BLOCKS, THREADS>>>(currentBoards, boardCount, emptySpaces, emptySpacesCounts, finished, gpuResult);

    executionEnd = clock();

    executionTime = ((float)(executionEnd - executionStart)) / (CLOCKS_PER_SEC / 1000);

    copyingStart = clock();
    gpuErrorCheck(hipMemcpy(result, gpuResult, BOARD_SIZE * BOARD_SIZE * sizeof(int), hipMemcpyDeviceToHost));
    copyingEnd = clock();
    dataCopyingTime += ((float)(copyingEnd - copyingStart)) / (CLOCKS_PER_SEC / 1000);

    std::cout << "Copying data took: " << dataCopyingTime << " ms\n";

    gpuErrorCheck(hipFree(nextBoards));
    gpuErrorCheck(hipFree(currentBoards));
    gpuErrorCheck(hipFree(boardIdx));
    gpuErrorCheck(hipFree(finished));
    gpuErrorCheck(hipFree(gpuResult));
    gpuErrorCheck(hipFree(emptySpaces));
    gpuErrorCheck(hipFree(emptySpacesCounts));

    return executionTime;
}
