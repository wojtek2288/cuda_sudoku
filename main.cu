﻿#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#include <iostream>

#include "io.h"
#include "defines.h"
#include "cpu.h"
#include "gpu.cuh"

void printSudoku(int *board)
{
    for (int i = 0; i < BOARD_SIZE; i++)
    {
        if (i % BOX_SIZE == 0)
        {
            std::cout << "-------------------------\n";
        }

        for (int j = 0; j < BOARD_SIZE; j++)
        {
            if (j % BOX_SIZE == 0)
            {
                std::cout << "| ";
            }

            std::cout << board[i * BOARD_SIZE + j] << " ";
        }

        std::cout << "|\n";
    }
    std::cout << "-------------------------\n";
}

void checkCpu(const int *sudoku, int *result)
{
    float timeTaken = solveWithCpu(sudoku, result);

    std::cout << "CPU solution: \n";
    printSudoku(result);
    std::cout << "Time taken for the cpu is: " << timeTaken << " ms\n";
}

void checkGpu(int *sudoku, int *result)
{
    float timeTaken = solveWithGpu(sudoku, result);

    std::cout << "GPU solution: \n";
    printSudoku(result);
    std::cout << "Time taken for the gpu is: " << timeTaken << " ms\n";
}

int main(int argc, char *argv[])
{
    int sudoku[BOARD_SIZE * BOARD_SIZE];
    int cpuResult[BOARD_SIZE * BOARD_SIZE];
    int gpuResult[BOARD_SIZE * BOARD_SIZE];

    FILE *inputFile;

    if (argc != 2)
    {
        std::cout << "Specify file with sudoku board";
        return EXIT_FAILURE;
    }

    if ((inputFile = fopen(argv[1], "r")) == NULL)
    {
        std::cout << "Could not open file: " << argv[1];
        return EXIT_FAILURE;
    }

    readSudokuFromFile(inputFile, sudoku);

    checkCpu(sudoku, cpuResult);
    std::cout << '\n';
    checkGpu(sudoku, gpuResult);
}